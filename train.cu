#include "hip/hip_runtime.h"
#include	<hip/hip_runtime.h>
#include	<hipblas.h>
#include	<sys/mman.h>
#include	<sys/stat.h>
#include	<sys/time.h>
#include	<algorithm>
#include	<iostream>
#include	<stdint.h>
#include	<unistd.h>
#include	<cstdlib>
#include	<fcntl.h>
#include	<vector>
#include	<cfloat>
using	namespace	std;
#define	tiger_beta	0.03125f
hipblasHandle_t	handle;
float	eta;
uint64_t	prng=time(NULL);
static inline uint64_t wyrand(uint64_t	*seed){	*seed+=0xa0761d6478bd642full;	uint64_t  see1=*seed^0xe7037ed1a0b428dbull;	see1*=(see1>>32)|(see1<<32);	return	(*seed*((*seed>>32)|(*seed<<32)))^((see1>>32)|(see1<<32));	}
static inline float wy2gau(uint64_t r){	const float	_wynorm=1.0/(1ull<<20);	return ((r&0x1fffff)+((r>>21)&0x1fffff)+((r>>42)&0x1fffff))*_wynorm-3.0f;	}
static inline double wy2u01(uint64_t r){	const double _wynorm=1.0/(1ull<<52);	return (r>>12)*_wynorm;	}
__device__ void _wymum(uint64_t *A,	uint64_t *B){	uint64_t	hh=(*A>>32)*(*B>>32), hl=(*A>>32)*(uint32_t)*B, lh=(uint32_t)*A*(*B>>32), ll=(uint64_t)(uint32_t)*A*(uint32_t)*B;	*A=((hl>>32)|(hl<<32))^hh;	*B=((lh>>32)|(lh<<32))^ll;	}
__device__	uint64_t	_wyhash64(uint64_t	A,	uint64_t	B){	A^=0xa0761d6478bd642full;	B^=0xa0761d6478bd642full;	_wymum(&A,&B);	A^=0xa0761d6478bd642full;	B^=0xa0761d6478bd642full;	_wymum(&A,&B);	return	A^B;	}
__global__	void	_tiger(float	*w, float	*m,	float	lr){	
	unsigned	i=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*w4=(float4*)(w+i),*m4=(float4*)(m+i);
	*w4=make_float4(w4->x-lr*((m4->x>0)-0.5f),w4->y-lr*((m4->y>0)-0.5f),w4->z-lr*((m4->z>0)-0.5f),w4->w-lr*((m4->w>0)-0.5f));
}
struct	bfloat8{	__hip_bfloat162	x,y,z,w;	};
struct	float8x{	float2	x,y,z,w;	};
__global__	void	_quant(float	*inp, __hip_bfloat16	*out){	
	unsigned	i=(blockIdx.x*blockDim.x+threadIdx.x)<<3;
	float8x	*i4=(float8x*)(inp+i);	bfloat8	o8;
	o8.x=__float22bfloat162_rn(i4->x);	o8.y=__float22bfloat162_rn(i4->y);	o8.z=__float22bfloat162_rn(i4->z);	o8.w=__float22bfloat162_rn(i4->w);
	*(bfloat8*)(out+i)=o8;
}
template<unsigned	N>
struct	Data16{
	__hip_bfloat16	*data;
	Data16(){	hipMallocManaged(&data,	N*sizeof(__hip_bfloat16));	}
	~Data16(){	hipFree(data);	}
};
__global__	void	_s16(unsigned	N,	float	*w, __hip_bfloat16	*g){	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x;	if(id<N)	g[id]=__float2bfloat16(w[id]);	}
__global__	void	_l16(unsigned	N,	float	*w, __hip_bfloat16	*g){	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x;	if(id<N)	w[id]=__bfloat162float(g[id]);	}
template<unsigned	N>
struct	Data{
	float	*data;
	Data(){	hipMallocManaged(&data,	N*sizeof(float));	}
	~Data(){	hipFree(data);	}
	void	save(FILE	*F){
		Data16<N>	tmp;	
		_s16<<<(N+15)/16,16>>>(N,data,tmp.data);	
		hipDeviceSynchronize();	
		fwrite(tmp.data,N*2,1,F);	
	}
	void	load(FILE	*F){
		Data16<N>	tmp;	
		if(fread(tmp.data,N*2,1,F)!=1){	return;	}	
		_l16<<<(N+15)/16,16>>>(N,data,tmp.data);	
		hipDeviceSynchronize();	
	}
	unsigned	size(void){	return	N;	}
	void	zero(void){	hipMemset(data,	0,	N*sizeof(float));	}
	void	rand(float	norm=1){	for(unsigned	i=0;	i<N;	i++)	data[i]=norm*wy2gau(wyrand(&prng));	}
	float	norm(void){	float	n;	hipblasSnrm2(handle,N,data,1,&n);	hipDeviceSynchronize();	return	n/sqrtf(N);	}
};
template<unsigned	R0,	unsigned	R1,	unsigned	C>
struct	linear{
	Data16<R0*R1>	weq;
	Data<R0*R1>	wei,wem;
	Data16<R0*C>	inq;
	Data16<R1*C>	giq;
	Data<R1*C>	out;
	linear(){	wei.rand(1/sqrtf(R0));	wem.zero();	}
	void	save(FILE	*F){	wei.save(F);	}
	void	load(FILE	*F){	wei.load(F);	}
	unsigned	size(void){	return	wei.size();	}
	uint64_t	flop(void){	return	6ull*R1*C*R0;	}
	void	fw(Data<R0*C>	&inp){
		float	alf=1/sqrtf(R0),	bet=0;
		_quant<<<R0*R1/8/16,16>>>(wei.data,weq.data);	_quant<<<R0*C/8/16,16>>>(inp.data,inq.data);
		hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,R1,C,R0,&alf,weq.data,HIP_R_16BF,R0,inq.data,HIP_R_16BF,R0,&bet,out.data,HIP_R_32F,R1,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
	}
	void	bk(Data<R0*C>	&inp,	Data<R1*C>	&gin,	Data<R0*C>	&gra,	bool	accumulate=false){
		float	alf=1/sqrtf(R0),	alf1=tiger_beta/sqrtf(R0*C),	bet=1-tiger_beta,	bet1=accumulate;
		_quant<<<R1*C/8/16,16>>>(gin.data,giq.data);
		hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,R0,R1,C,&alf1,inq.data,HIP_R_16BF,R0,giq.data,HIP_R_16BF,R1,&bet,wem.data,HIP_R_32F,R0,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);		
		hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,R0,C,R1,&alf,weq.data,HIP_R_16BF,R0,giq.data,HIP_R_16BF,R1,&bet1,gra.data,HIP_R_32F,R0,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		_tiger<<<R0*R1/4/16,16>>>(wei.data,wem.data,eta);
	}
};
__global__	void	_layernormf(unsigned	R,	unsigned	C,	unsigned	H,	float	*inp,	float	*norm){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	r=R/H;
	float	*in=inp+id*r,	sum=0,	nor=0;
	for(unsigned	i=0;	i<r;	i+=4){	float4*	t=(float4*)(in+i);	sum+=t->x+t->y+t->z+t->w;	nor+=t->x*t->x+t->y*t->y+t->z*t->z+t->w*t->w;	}
	sum/=r;	nor=fmaxf(nor-sum*sum*r,1e-18f);	norm[id]=nor;	nor=sqrtf(r/nor);
	for(unsigned	i=0;	i<r;	i+=4){	float4	*t=(float4*)(in+i);	*t=make_float4((t->x-sum)*nor,(t->y-sum)*nor,(t->z-sum)*nor,(t->w-sum)*nor);	}
}
__global__	void	_layernormb(unsigned	R,	unsigned	C,	unsigned	H,	float	*inp,	float	*gin,	float	*norm){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	r=R/H;
	float	*gi=gin+id*r,	*ou=inp+id*r,	mg=0,	 sgi=0,sou=0,	s=sqrtf(r/norm[id]),	sum=0;
	for(unsigned	i=0;	i<r;	i+=4){	float4	*g=(float4*)(gi+i),	*o=(float4*)(ou+i);	sgi+=g->x+g->y+g->z+g->w;	sou+=o->x+o->y+o->z+o->w;	mg+=g->x*o->x+g->y*o->y+g->z*o->z+g->w*o->w;	}
	mg/=norm[id]*s;	sum=(s*sgi-mg*sou)/r;
	for(unsigned	i=0;	i<r;	i+=4){	float4	*g=(float4*)(gi+i),	*o=(float4*)(ou+i);	*g=make_float4(s*g->x-mg*o->x-sum,s*g->y-mg*o->y-sum,s*g->z-mg*o->z-sum,s*g->w-mg*o->w-sum);	}
}
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	layernorm{
	Data<C*H>	nor;
	void	fw(Data<R*C>	&inp){	_layernormf<<<C*H/16,16>>>(R,C,H,inp.data,nor.data);	}
	void	bk(Data<R*C>	&inp,	Data<R*C>	&gin){	_layernormb<<<C*H/16,16>>>(R,C,H,inp.data,gin.data,nor.data);	}
};
__global__	void	_softmaxf(unsigned	R,	float	*inp){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x;
	float	*p=inp+id*R,	sum=0,	ma=-FLT_MAX;
	for(unsigned	i=0;    i<R;    i++)	ma=fmaxf(p[i],ma);
	for(unsigned	i=0;	i<R;	i++)	sum+=(p[i]=expf(p[i]-ma));
	for(unsigned	i=0;	i<R;	i++)	p[i]/=sum;
}
__global__	void	_sexyfp(unsigned	C,	float	*att,	float	*pe){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	c=id%C,	h=id/C;
	float	*a=att+id*C,	*p=pe+h*C+c;
	for(unsigned	i=0;	i<=c;	i++)	a[i]=expf(*(p-i)+a[i]);
	for(unsigned	i=c+1;	i<C;	i++)	a[i]=0;
}
__global__	void	_sexyfsuv(unsigned	R,	float	*u,	float	*v,	float	*out){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+(id/R)*4*R+3*R+(id%R)),	*v4=(float4*)(v+id),	*o4=(float4*)(out+id);
	*o4=make_float4(u4->x*v4->x,u4->y*v4->y,u4->z*v4->z,u4->w*v4->w);
}
__global__	void	_sexybsuv(unsigned	R,	float	*u,	float	*v,	float	*gin,	float	*gx){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+(id/R)*4*R+3*R+(id%R)),	*v4=(float4*)(v+id),	*g4=(float4*)(gin+id),	*x4=(float4*)(gx+(id/R)*4*R+3*R+(id%R));
	*x4=make_float4(v4->x*g4->x,v4->y*g4->y,v4->z*g4->z,v4->w*g4->w);
	*v4=make_float4(u4->x*g4->x,u4->y*g4->y,u4->z*g4->z,u4->w*g4->w);
}
__global__	void	_sexyba(float	*gin,	float	*att){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*g4=(float4*)(gin+id),	*a4=(float4*)(att+id);
	*g4=make_float4(a4->x*g4->x,a4->y*g4->y,a4->z*g4->z,a4->w*g4->w);
}
__global__	void	_sexybp(unsigned	R,	unsigned	C,	float	*a,	float	*pe,	float	*pm,	float	eta){
	unsigned  id=blockIdx.x*blockDim.x+threadIdx.x,	h=id/C,	c=id%C;	float	s=0,*p=a+h*C*C;
	for(unsigned	i=c;	i<C;	i++)	s+=p[i*C+(i-c)];
	s/=sqrtf((C-c)*R);	pm[id]+=tiger_beta*(s-pm[id]);	pe[id]-=eta*((pm[id]>0)-0.5f);
}
__global__	void	_sexyadd(unsigned	R,	unsigned	H,	float	*inp,	float	*out){
	unsigned  id=(blockIdx.x*blockDim.x+threadIdx.x)<<2,c=id/R,r=id%R;	float4	s={},*o4=(float4*)(out+id);
	for(unsigned	h=0;	h<H;	h++){	float4*	i4=(float4*)(inp+c*R*H+h*R+r);	s=make_float4(s.x+i4->x,s.y+i4->y,s.z+i4->z,s.w+i4->w);	}
	*o4=make_float4(s.x+o4->x,s.y+o4->y,s.z+o4->z,s.w+o4->w);
}
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	sexy{
	static	Data<C*C*H>	da;
	static	Data<R*C>	gi;
	static	Data<4*R*C>	gx;
	static	Data16<R*C>	vaq;
	Data16<4*R*C>	xq;
	Data16<C*C*H>	atq;
	Data<C*C*H>	at;
	Data<R*C>	va,tmp;
	layernorm<R,C,H>	n1;
	layernorm<4*R,C,4*H>	n4;
	Data<H*C>	pe,pm;
	linear<R,4*R,C>	x;
	linear<R,R,C>	o;
	Data<R*C>	&out=o.out;
	sexy(){	pe.zero();	pm.zero();	}
	void	save(FILE	*F){	pe.save(F);	x.save(F);	o.save(F);	}
	void	load(FILE	*F){	pe.load(F);	x.load(F);	o.load(F);	}
	unsigned	size(void){	return	pe.size()+x.size()+o.size();	}
	uint64_t	flop(void){	return	x.flop()+o.flop()+12ull*C*C*R;	}
	void	fw(Data<R*C>	&inp){
		float	alf=1/sqrtf(R/H),alf1=1,bet=0;
		x.fw(inp);	n4.fw(x.out);	_quant<<<4*R*C/8/16,16>>>(x.out.data,xq.data);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,C,C,R/H,&alf,xq.data,HIP_R_16BF,4*R,R/H,xq.data+R,HIP_R_16BF,4*R,R/H,&bet,at.data,HIP_R_32F,C,C*C,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		_sexyfp<<<C*H/16,16>>>(C,at.data,pe.data);	_quant<<<C*C*H/8/16,16>>>(at.data,atq.data);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,R/H,C,C,&alf1,xq.data+2*R,HIP_R_16BF,4*R,R/H,atq.data,HIP_R_16BF,C,C*C,&bet,va.data,HIP_R_32F,R,R/H,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		_sexyfsuv<<<R*C/4/16,16>>>(R,x.out.data,va.data,tmp.data);	n1.fw(tmp);	o.fw(tmp);
		hipblasSaxpy(handle,R*C,&alf1,inp.data,1,out.data,1);
	}
	void	bk(Data<R*C>	&inp,	Data<R*C>	&gin,	Data<R*C> &gra){
		float	alf=1/sqrtf(R/H),alf1=1,bet=0;
		o.bk(tmp,gin,gi);	n1.bk(tmp,gi);	_sexybsuv<<<R*C/4/16,16>>>(R,x.out.data,va.data,gi.data,gx.data);	_quant<<<R*C/8/16,16>>>(va.data,vaq.data);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,C,C,R/H,&alf1,xq.data+2*R,HIP_R_16BF,4*R,R/H,vaq.data,HIP_R_16BF,R,R/H,&bet,da.data,HIP_R_32F,C,C*C,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,R/H,C,C,&alf1,vaq.data,HIP_R_16BF,R,R/H,atq.data,HIP_R_16BF,C,C*C,&bet,gx.data+2*R,HIP_R_32F,4*R,R/H,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		_sexyba<<<C*C*H/4/16,16>>>(da.data,at.data);	_sexybp<<<H*C/16,16>>>(R,C,da.data,pe.data,pm.data,eta);	_quant<<<C*C*H/8/16,16>>>(da.data,atq.data);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,R/H,C,C,&alf,xq.data+R,HIP_R_16BF,4*R,R/H,atq.data,HIP_R_16BF,C,C*C,&bet,gx.data,HIP_R_32F,4*R,R/H,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,R/H,C,C,&alf,xq.data,HIP_R_16BF,4*R,R/H,atq.data,HIP_R_16BF,C,C*C,&bet,gx.data+R,HIP_R_32F,4*R,R/H,H,HIPBLAS_COMPUTE_32F,HIPBLAS_GEMM_DEFAULT);
		n4.bk(x.out,gx);	x.bk(inp,gx,gra);	hipblasSaxpy(handle,R*C,&alf1,gin.data,1,gra.data,1);
	}
};
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<C*C*H>	sexy<R,C,H>::da;
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<R*C>	sexy<R,C,H>::gi;
template<unsigned	R,	unsigned	C,	unsigned	H>
Data16<R*C>	sexy<R,C,H>::vaq;
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<4*R*C>	sexy<R,C,H>::gx;
__global__	void	_selffsuv(unsigned	S,	float	*u,	float	*out){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+(id/S)*2*S+(id%S)),	*v4=(float4*)(u+(id/S)*2*S+S+(id%S)),	*o4=(float4*)(out+id);
	*o4=make_float4(u4->x*v4->x,u4->y*v4->y,u4->z*v4->z,u4->w*v4->w);
}
__global__	void	_selfbsuv(unsigned	S,	float	*u,	float	*gin,	float	*d){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4  *u4=(float4*)(u+(id/S)*2*S+(id%S)),	*v4=(float4*)(u+(id/S)*2*S+S+(id%S)),	*g4=(float4*)(gin+id),	*p4=(float4*)(d+(id/S)*2*S+(id%S)),	*q4=(float4*)(d+(id/S)*2*S+S+(id%S));
	*p4=make_float4(v4->x*g4->x,v4->y*g4->y,v4->z*g4->z,v4->w*g4->w);	*q4=make_float4(u4->x*g4->x,u4->y*g4->y,u4->z*g4->z,u4->w*g4->w);
}
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	self{
	static	Data<R*C>	gi;
	static	Data<2*R*C>	du;
	Data<R*C>	tmp;
	layernorm<2*R,C,2*H>	n2;
	layernorm<R,C,H>	n1;
	linear<R,2*R,C>	u;
	linear<R,R,C>	o;
	Data<R*C>	&out=o.out;
	void	save(FILE	*F){	u.save(F);	o.save(F);	}
	void	load(FILE	*F){	u.load(F);	o.load(F);	}
	unsigned	size(void){	return	u.size()+o.size();	}
	uint64_t	flop(void){	return	u.flop()+o.flop();	}
	void	fw(Data<R*C>	&inp){
		float	alf1=1;
		u.fw(inp);	n2.fw(u.out);
		_selffsuv<<<R*C/4/16,16>>>(R,u.out.data,tmp.data);	n1.fw(tmp);	o.fw(tmp);
		hipblasSaxpy(handle,R*C,&alf1,inp.data,1,out.data,1);
	}
	void	bk(Data<R*C>	&inp,	Data<R*C>	&gin,	Data<R*C> &gra){
		float	alf1=1;
		o.bk(tmp,gin,gi);	n1.bk(tmp,gi);
		_selfbsuv<<<R*C/4/16,16>>>(R,u.out.data,gi.data,du.data);	
		n2.bk(u.out,du);	u.bk(inp,du,gra);
		hipblasSaxpy(handle,R*C,&alf1,gin.data,1,gra.data,1);
	}
};
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<R*C>	self<R,C,H>::gi;
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<2*R*C>	self<R,C,H>::du;
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	wyGPT{
	self<R,C,H>	a;
	sexy<R,C,H>	b;
	self<R,C,H>	c;
	Data<R*C>	&out=c.out;
	void	save(FILE	*F){	a.save(F);	b.save(F);	c.save(F);	}
	void	load(FILE	*F){	a.load(F);	b.load(F);	c.load(F);	}
	unsigned	size(void){	return	a.size()+b.size()+c.size();	}
	uint64_t	flop(void){	return	a.flop()+b.flop()+c.flop();	}
	void	fw(Data<R*C>	&inp){
		a.fw(inp);
		b.fw(a.out);
		c.fw(b.out);
	}
	void	bk(Data<R*C>	&inp,	Data<R*C>	&gin,	Data<R*C> &gra){
		c.bk(b.out,gin,gra);
		b.bk(a.out,gra,gin);
		a.bk(inp,gin,gra);
	}
};
__global__	void	_emb(unsigned	R,	unsigned	C,	uint8_t	*inp,	float	*out){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	r=id%R,	c=(id/R)%C;	
	out[id]=(_wyhash64(inp[c],r)&1)*2-1.0f;
}
__global__	void	dlossf(unsigned	C,	unsigned	O,	float	*a,	uint8_t	*x,	float	*y){
	float	loss=0;
	for(unsigned	i=0;	i<C;	i++){
		loss-=logf(fmaxf(a[i*O+x[i+1]],FLT_MIN));
		a[i*O+x[i+1]]-=1;
	}
	*y=loss;
}
template<unsigned	C,	unsigned	E,	unsigned	D,	unsigned	H,	unsigned	O>
struct	Neanderthal{
private:
	float	*ret;
	uint8_t	*data;
	Data<E*C>	n0g,trag[2];
public:
	uint64_t	srng=time(NULL);
	Data<E*C>	emb;
	wyGPT<E,C,H>	tra[D];
	layernorm<E,C,1>	n1;
	linear<E,O,C>	ou;
	Neanderthal(){	hipMallocManaged(&data,	C+1);	hipMallocManaged(&ret,	sizeof(float));	}
	~Neanderthal(){	hipFree(data);	hipFree(ret);	}
	bool	save(const	char	*F){
		FILE	*f=fopen(F,"wb");	if(f==NULL)	return	false;
		unsigned	x;
		x=C;	fwrite(&x,4,1,f);
		x=E;	fwrite(&x,4,1,f);
		x=D;	fwrite(&x,4,1,f);
		x=H;	fwrite(&x,4,1,f);
		x=O;	fwrite(&x,4,1,f);		
		for(unsigned	i=0;	i<D;	i++)	tra[i].save(f);
		ou.save(f);	fclose(f);	return	true;
	}
	bool	load(const	char	*F){
		FILE	*f=fopen(F,"rb");	if(f==NULL)	return	false;
		unsigned	x;
		if(fread(&x,4,1,f)!=1||x!=C)	fprintf(stderr,"C=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=E)	fprintf(stderr,"E=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=D)	fprintf(stderr,"D=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=H)	fprintf(stderr,"H=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=O)	fprintf(stderr,"O=%u\n",x);	
		for(unsigned	i=0;	i<D;	i++)	tra[i].load(f);
		ou.load(f);	fclose(f);	return	true;
	}
	unsigned	size(void){	return	tra[0].size()*D+ou.size();	}
	float	train(uint8_t	*text,	uint64_t	len){
		hipMemcpy(data,text+(wyrand(&srng)%(len-C)),C+1,hipMemcpyHostToDevice);
		_emb<<<E*C/16,16>>>(E,C,data,emb.data);
		for(unsigned	d=0;	d<D;	d++)	tra[d].fw(d?tra[d-1].out:emb);
		n1.fw(tra[D-1].out);	ou.fw(tra[D-1].out);
		_softmaxf<<<C/16,16>>>(O,ou.out.data);
		dlossf<<<1,1>>>(C,O,ou.out.data,data,ret);
		ou.bk(tra[D-1].out,ou.out,n0g);	n1.bk(tra[D-1].out,n0g);
		for(unsigned	d=D-1;	d<D;	d--)	tra[d].bk(d?tra[d-1].out:emb,d<D-1?trag[(d+1)%2]:n0g,trag[d%2]);
		hipDeviceSynchronize();	return	*ret;
	}
};
#include	"config"
using	namespace	std;
Neanderthal<context,embed,depth,heads,voca>	model;
void	document(void){
	cerr<<"usage:	train [options] input1.txt [input2.txt input3.txt...]\n";
	cerr<<"\t-i:	input model=NULL\n";
	cerr<<"\t-o:	output model=model\n";
	cerr<<"\t-s:	trained sample=0\n";
	cerr<<"\t-b:	benchmark only=off\n";
	exit(0);
}
struct	Dataset{
	string	name;
	uint8_t	*ptr;
	int	fd;
	struct	stat	sb;
	double	weight;
};
int	main(int	ac,	char	**av){
	hipblasCreate(&handle);	string	in,out="model";	int	opt,bench=0;	uint64_t	training=0;
	while((opt=getopt(ac,	av,	"i:o:s:b"))>=0){
		switch(opt){
		case	'i':	in=optarg;	break;
		case	'o':	out=optarg;	break;
		case	's':{	training=atoi(optarg);	training<<=20;	}	break;
		case	'b':	bench=1;	model.srng=0;	break;
		default:	document();
		}
	}
	if(ac<optind+1){	document();	return	0;	}
	vector<Dataset>	ds;	ds.resize(ac-optind);	double	sum_weight=0;
	for(int	i=optind;	i<ac;	i++){
		int	j=i-optind;
		ds[j].name=av[i];	ds[j].fd=open(av[i],	O_RDONLY);	fstat(ds[j].fd,	&ds[j].sb);
		ds[j].ptr=(uint8_t*)mmap(NULL,	ds[j].sb.st_size,	PROT_READ,	MAP_SHARED,	ds[j].fd,	0);
		sum_weight+=(ds[j].weight=1);
		cerr<<av[i]<<'\t'<<ds[j].sb.st_size/1024.0f/1024<<'\t'<<ds[j].weight<<'\n';
	}
	cerr.precision(4);	cerr.setf(ios::fixed);
	double	loss0=FLT_MAX/2,	loss;	timeval	beg,	end;	
	size_t	para=model.size();	cerr<<"para\t"<<para<<'\n';	
	if(in.size())	model.load(in.c_str());
	for(;;){
		loss=0;	gettimeofday(&beg,NULL);	vector<double>	vl(ds.size()),vn(ds.size());	
		for(size_t	i=0;	i<fullbatch;	i++){
			eta=2.0f/sqrtf(log1pf(para)*para+training);	training+=context;
			double	ran=wy2u01(wyrand(&prng))*sum_weight,sum=0;
			size_t	r=ds.size()-1;
			for(size_t	j=0;	j<ds.size();	j++){	
				sum+=ds[j].weight;
				if(sum>=ran){	r=j;	break;	}
			}
			double	l=model.train(ds[r].ptr,ds[r].sb.st_size);
			loss+=l;	vl[r]+=l;	vn[r]+=context;
		}
		loss/=context*fullbatch;
		if(!bench){	if(loss<loss0+0.02)	model.save(out.c_str());	else	break;	}
		loss0=loss;	gettimeofday(&end,NULL);
		cerr<<(training>>20);
		for(size_t	i=0;	i<ds.size();	i++)	cerr<<'\t'<<vl[i]/vn[i];
		cerr<<'\t'<<(end.tv_sec-beg.tv_sec+1e-6*(end.tv_usec-beg.tv_usec))<<'\n';
	}
	for(int	j=0;	j<ds.size();	j++){	munmap(ds[j].ptr,ds[j].sb.st_size);	close(ds[j].fd);	}
	hipblasDestroy(handle);
	return	0;
}
