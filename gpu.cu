#include "hip/hip_runtime.h"
#include	<hip/hip_runtime.h>
#include	<unordered_map>
#include	<hipblas.h>
#include	<sys/time.h>
#include	<algorithm>
#include	<iostream>
#include	<unistd.h>
#include	<sstream>
#include	<cstdint>
#include	<cfloat>
#include	<cstdio>
#include	<vector>
using	namespace	std;
hipblasHandle_t	handle;
uint64_t	prng=time(NULL);
static inline uint64_t wyrand(uint64_t	*seed){	*seed+=0xa0761d6478bd642full;	uint64_t  see1=*seed^0xe7037ed1a0b428dbull;	see1*=(see1>>32)|(see1<<32);	return	(*seed*((*seed>>32)|(*seed<<32)))^((see1>>32)|(see1<<32));	}
static inline double wy2u01(uint64_t r){	const double _wynorm=1.0/(1ull<<52);	return (r>>12)*_wynorm;	}
void _wymum(uint64_t *A,	uint64_t *B){	uint64_t	hh=(*A>>32)*(*B>>32), hl=(*A>>32)*(uint32_t)*B, lh=(uint32_t)*A*(*B>>32), ll=(uint64_t)(uint32_t)*A*(uint32_t)*B;	*A=((hl>>32)|(hl<<32))^hh;	*B=((lh>>32)|(lh<<32))^ll;	}
uint64_t	_wyhash64(uint64_t	A,	uint64_t	B){	A^=0xa0761d6478bd642full;	B^=0xa0761d6478bd642full;	_wymum(&A,&B);	A^=0xa0761d6478bd642full;	B^=0xa0761d6478bd642full;	_wymum(&A,&B);	return	A^B;	}
template<unsigned	N>
struct	Data16{
	__hip_bfloat16	*data;
	Data16(){	hipMallocManaged(&data,	N*sizeof(__hip_bfloat16));	}
	~Data16(){	hipFree(data);	}
	void	zero(void){	hipMemset(data,	0,	N*sizeof(__hip_bfloat16));	}
	void	load(FILE	*F){	if(fread(data,N*2,1,F)!=1)	return;	}	
};
__global__	void	_s16(unsigned	N,	float	*w, __hip_bfloat16	*g){	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x;	if(id<N)	g[id]=__float2bfloat16(w[id]);	}
__global__	void	_l16(unsigned	N,	float	*w, __hip_bfloat16	*g){	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x;	if(id<N)	w[id]=__bfloat162float(g[id]);	}
template<unsigned	N>
struct	Data{
	static	Data16<N>	tmp;
	float	*data;
	Data(){	hipMallocManaged(&data,	N*sizeof(float));	}
	~Data(){	hipFree(data);	}
	void	zero(void){	hipMemset(data,	0,	N*sizeof(float));	}
	void	load(FILE	*F){	if(fread(tmp.data,N*2,1,F)!=1){	return;	}	_l16<<<(N+15)/16,16>>>(N,data,tmp.data);	hipDeviceSynchronize();	}
};
template<unsigned	N>
Data16<N>	Data<N>::tmp;
template<unsigned	R0,	unsigned	R1>
struct	linear{
	Data16<R0*R1>	wei;
	Data<R1>	out;
	void	load(FILE	*F){	wei.load(F);	}
	void	fw(Data<R0>	&inp){
		float	alf=1/sqrtf(R0),	bet=0;
		_s16<<<R0/16,16>>>(R0,inp.data,inp.tmp.data);
		cublasTSSgemvStridedBatched(handle,HIPBLAS_OP_T,R0,R1,&alf,wei.data,R0,0,inp.tmp.data,1,0,&bet,out.data,1,0,1);
	}
};
__global__	void	_layernorm(unsigned	R,	float	*inp,	unsigned	H){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	r=R/H;
	float	sum=0,	nor=0,	*in=inp+id*r;	
	for(unsigned	i=0;	i<r;	i+=4){	float4*	t=(float4*)(in+i);	sum+=t->x+t->y+t->z+t->w;	nor+=t->x*t->x+t->y*t->y+t->z*t->z+t->w*t->w;	}
	sum/=r;	nor=sqrtf(r/fmaxf(nor-sum*sum*r,1e-18f));
	for(unsigned	i=0;	i<r;	i+=4){	float4	*t=(float4*)(in+i);	*t=make_float4((t->x-sum)*nor,(t->y-sum)*nor,(t->z-sum)*nor,(t->w-sum)*nor);	}
}
void	softmax(unsigned	R,	float	*inp){
	float	sum=0,	ma=-FLT_MAX;
	for(unsigned	i=0;    i<R;    i++)	ma=fmaxf(inp[i],ma);
	for(unsigned	i=0;	i<R;	i++)	sum+=(inp[i]=expf(inp[i]-ma));
	for(unsigned	i=0;	i<R;	i++)	inp[i]/=sum;
}
__global__	void	_sexyfp(unsigned	C,	unsigned	para,	unsigned	col,	float	*att,	float	*pe){
	unsigned	id=blockIdx.x*blockDim.x+threadIdx.x,	j=id%C,	h=id/C,	i=(j+1+col)%C;
	if(j<para)	att[h*C+i]=0;
	else	att[h*C+i]=expf(pe[h*C+C-1-j]+att[h*C+i]);
}
__global__	void	_sexyfsuv(float	*u,	float	*v){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+id),	*v4=(float4*)(v+id);
	*v4=make_float4(u4->x*v4->x,u4->y*v4->y,u4->z*v4->z,u4->w*v4->w);
}
__global__	void	_sexyadd(float	*u,	float	*v){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+id),	*v4=(float4*)(v+id);
	*u4=make_float4(u4->x+v4->x,u4->y+v4->y,u4->z+v4->z,u4->w+v4->w);
}
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	sexy{
	static	Data<R>	va;
	static	Data<H*C>	a;
	Data16<R*C>	k0,k1;
	Data<H*C>	pe;
	linear<R,4*R>	x;
	linear<R,R>	o;
	Data<R>	&out=o.out;
	sexy(){	k0.zero();	k1.zero();	}
	void	load(FILE* F){	 pe.load(F); x.load(F);	o.load(F);	}
	void	fw(Data<R>	&inp,	unsigned	col,	unsigned	para){
		x.fw(inp);	_layernorm<<<4*H,1>>>(4*R,x.out.data,4*H);
		_s16<<<R/16,16>>>(R,x.out.data,k0.data+col*R);
		_s16<<<R/16,16>>>(R,x.out.data+R,x.out.tmp.data+R);
		_s16<<<R/16,16>>>(R,x.out.data+2*R,k1.data+col*R);
		float	alf=1/sqrtf(R/H),	alf1=1,bet=0;
		cublasTSSgemvStridedBatched(handle,HIPBLAS_OP_T,R/H,C,&alf,k0.data,R,R/H,x.out.tmp.data+R,1,R/H,&bet,a.data,1,C,H);
		_sexyfp<<<C*H/16,16>>>(C,para,col,a.data,pe.data);
		_s16<<<H*C/16,16>>>(H*C,a.data,a.tmp.data);
		cublasTSSgemvStridedBatched(handle,HIPBLAS_OP_N,R/H,C,&alf1,k1.data,R,R/H,a.tmp.data,1,C,&bet,va.data,1,R/H,H);
		_sexyfsuv<<<R/4/4,4>>>(x.out.data+3*R,va.data);
		_layernorm<<<H,1>>>(R,va.data,H);	o.fw(va);
		_sexyadd<<<R/4/4,4>>>(o.out.data,inp.data);
	}
};
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<R>	sexy<R,C,H>::va;
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<H*C>	sexy<R,C,H>::a;
__global__	void	_selffsuv(unsigned	S,	float	*u,	float	*o){
	unsigned	id=(blockIdx.x*blockDim.x+threadIdx.x)<<2;
	float4	*u4=(float4*)(u+id),	*v4=(float4*)(u+S+id),	*o4=(float4*)(o+id);
	*o4=make_float4(u4->x*v4->x,u4->y*v4->y,u4->z*v4->z,u4->w*v4->w);
}
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	self{
	static	Data<R>	tmp;
	linear<R,2*R>	u;
	linear<R,R>	o;
	Data<R>	&out=o.out;
	void	load(FILE* F){	 u.load(F);	o.load(F);	}
	void	fw(Data<R>	&inp){
		u.fw(inp);	_layernorm<<<2*H,1>>>(2*R,u.out.data,2*H);
		_selffsuv<<<R/4/4,4>>>(R,u.out.data,tmp.data);
		_layernorm<<<H,1>>>(R,tmp.data,H);	o.fw(tmp);
		_sexyadd<<<R/4/4,4>>>(o.out.data,inp.data);
	}
};
template<unsigned	R,	unsigned	C,	unsigned	H>
Data<R>	self<R,C,H>::tmp;
template<unsigned	R,	unsigned	C,	unsigned	H>
struct	wyGPT{
	self<R,C,H>	a;
	sexy<R,C,H>	b;
	self<R,C,H>	c;
	Data<R>	&out=c.out;
	void	load(FILE* F){	 a.load(F);	b.load(F);	c.load(F);	}
	void	fw(Data<R>	&inp,	unsigned	col,	unsigned	para){
		a.fw(inp);
		b.fw(a.out,col,para);
		c.fw(b.out);
	}
};
template<unsigned	C,	unsigned	E,	unsigned	D,	unsigned	H,	unsigned	O>
struct	Neanderthal{
	unsigned	curr=0;
	Data<E>	emb;
	wyGPT<E,C,H>	tra[D];
	linear<E,O>	out;
	float	vs[O];
	bool	load(const	char	*F){
		FILE* f=fopen(F, "rb");
		if(f==NULL)	return	false;
		unsigned	x;		
		if(fread(&x,4,1,f)!=1||x!=C)	fprintf(stderr,"C=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=E)	fprintf(stderr,"E=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=D)	fprintf(stderr,"D=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=H)	fprintf(stderr,"H=%u\n",x);
		if(fread(&x,4,1,f)!=1||x!=O)	fprintf(stderr,"O=%u\n",x);	
		for(unsigned i=0; i<D; i++)	tra[i].load(f);
		out.load(f);	fclose(f);
		return	true;
	}
	uint8_t	sample(uint8_t	*x,	uint8_t	*p){
		unsigned	para=p+C-1>=x?p+C-1-x:0;
		for(unsigned	r=0;	r<E;	r++)	emb.data[r]=(_wyhash64(*x,r)&1)*2-1.0f;
		for(unsigned	d=0;	d<D;	d++)	tra[d].fw(d?tra[d-1].out:emb,curr,para);
		_layernorm<<<1,1>>>(E,tra[D-1].out.data,1);	out.fw(tra[D-1].out);
		hipDeviceSynchronize();
		for(unsigned    i=0;    i<O;    i++)	out.out.data[i]=M_SQRT2*(out.out.data[i]-vs[i]);
		softmax(O,out.out.data);
		double	sum=0;	for(unsigned    i=0;    i<O;    i++)	sum+=(out.out.data[i]=fmaxf(out.out.data[i]-1.0f/O,0));
		double  ran=wy2u01(wyrand(&prng))*sum,  sum1=0; uint16_t        ret=0;
		for(size_t      i=0;    i<O;    i++){   sum1+=out.out.data[i];    if(sum1>=ran){  ret=i;  break;  }       }
		curr=(curr+1)%C;	return	ret;
	}
	string	generate(string	inp,	unsigned	n){
		if(!inp.size())	return	"";
		vector<uint8_t>	s;	uint8_t	c;
		for(unsigned	i=0;	i<inp.size()&&i<n;	i++){	
			s.push_back(inp[i]);	
			memset(vs,0,sizeof(float)*O);
			for(size_t	k=0;	k<s.size();	k++){
				unsigned	l=1;
				while(l<=k&&s[k-l]==s[s.size()-l])	l++;
				vs[s[k]]+=(expf(l/M_E)-1)/(s.size()-k);
			}
			c=sample(s.data()+s.size()-1,s.data());	
		}
		while(s.size()<n){	
			s.push_back(c);
			memset(vs,0,sizeof(float)*O);
			for(size_t	k=0;	k<s.size();	k++){
				unsigned	l=1;
				while(l<=k&&s[k-l]==s[s.size()-l])	l++;
				vs[s[k]]+=(expf(l/M_E)-1)/(s.size()-k);
			}
			c=sample(s.data()+s.size()-1,s.data());	
		}
		string	ret(s.begin(),s.end());
		return	ret;
	}
	float	probability(const	uint8_t	*x,	const	uint8_t	*p){
		unsigned	para=p+C-1>=x?p+C-1-x:0;
		for(unsigned	r=0;	r<E;	r++)	emb.data[r]=(_wyhash64(*x,r)&1)*2-1.0f;
		for(unsigned	d=0;	d<D;	d++)	tra[d].fw(d?tra[d-1].out:emb,curr,para);
		_layernorm<<<1,1>>>(E,tra[D-1].out.data,1);	out.fw(tra[D-1].out);
		hipDeviceSynchronize();	
		softmax(O,out.out.data);	curr=(curr+1)%C;	return	out.out.data[x[1]];
	}
	float	evaluate(string	inp){
		double	loss=0;
		for(unsigned	i=0;	i+1<inp.size();	i++)	
			loss-=logf(fmaxf(probability((uint8_t*)inp.data()+i,(uint8_t*)inp.data()),FLT_MIN));
		return	inp.size()<2?0:loss/(inp.size()-1);
	}
};
#include	"config"
int	main(int	ac,	char	**av){
	hipblasCreate(&handle);
	Neanderthal<context,embed,depth,heads,voca>	model;
	string	model_file="model";
	int	opt;
	while((opt=getopt(ac,	av,	"m:"))>=0){
		switch(opt){
		case	'm':	model_file=optarg;	break;
		}
	}
	if(!model.load(model_file.c_str())){	fprintf(stderr,"fail to load %s\n",model_file.c_str());	return	0;	}
	timeval	beg,end;	gettimeofday(&beg,NULL);
	cout<<model.generate(av[optind],context)<<'\n';	// the second parameter can be arbitary long
	gettimeofday(&end,NULL);
	cerr<<end.tv_sec-beg.tv_sec+1e-6*(end.tv_usec-beg.tv_usec)<<'\n';
	hipblasDestroy(handle);
	return	0;
}
